/***********************************************************************************************************************
 *
 *   CMPE 214 Final Project: Optimized Breadth First Search     
 *   Samir Mohammed & Boxiang Guo                             		    
 *
 **********************************************************************************************************************/

/***********************************************************************************************************************
 *
 *                                     		    I N C L U D E S
 *
 **********************************************************************************************************************/
// CUDA includes
#include "hip/hip_runtime.h"


#include <hip/hip_runtime_api.h>

// Standard includes
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

/***********************************************************************************************************************
 *
 *                                                   D E F I N E S
 *
 **********************************************************************************************************************/
#define NUMBER_OF_VERTICES 5

/***********************************************************************************************************************
 *
 *                                                  T Y P E D E F S
 *
 **********************************************************************************************************************/
typedef struct
{
	int index_of_first_adjacent_node_in_edge_array;    	
	int number_of_adjacent_nodes;    
} vertex;

/***********************************************************************************************************************
 *
 *                                     		     K E R N E L S
 *
 **********************************************************************************************************************/
__global__ void bfs_gpu (vertex *vertices, int *edges, bool *frontier, bool *visited, int *cost, bool *done)
{
	// generate global thread ID
	int gid = threadIdx.x + blockIdx.x * blockDim.x;

	// Loop thresholds
	int start, end;

	// Perform boundary check
	if (gid > NUMBER_OF_VERTICES)
	{
		*done = false;
	}
		
	// If vertex entry in frontier array is true and vertex has not been visited...
	if (frontier[gid] == true && visited[gid] == false)
	{
		// Print the order of the vertices in BFS
		printf("%d ", gid); 	

		// vertex has been visited
		frontier[gid] = false; // Remove vertex from frontier array
		visited[gid] = true; // Add vertex to visited array

		__syncthreads(); 

		// Initialize loop thresholds
		start = vertices[gid].index_of_first_adjacent_node_in_edge_array;
		end = start + vertices[gid].number_of_adjacent_nodes;

		// If neighbor has not been visited then add neighbor to frontier array
		for (int i = start; i < end; i++) 
		{
			int neighbor_id = edges[i];

			// If neighbor has not been visited...
			if (visited[neighbor_id] == false)
			{
				cost[neighbor_id] = cost[gid] + 1; // Update cost array
				frontier[neighbor_id] = true; // Add neighbor to frontier array
				*done = false;
			}
		}
	}
}

/***********************************************************************************************************************
 *
 *                                     			M A I N
 *
 **********************************************************************************************************************/
int main()
{
	vertex vertices[NUMBER_OF_VERTICES]; // Stores all vertices in graph
	int edges[NUMBER_OF_VERTICES]; // Stores all edges in graph

	bool done;
	
	vertex* device_vertices; // device_vertices stores the list of vertices
	int* device_edges; // device_edges stores the list of edges 
	bool* device_frontier; // device_frontier stores BFS frontier
	bool* device_visited; // device_visited stores visited vertices
	int* device_cost; // device_cost stores the mimimal number of edges from each vertex to source vertex
	bool* device_done;

	// Kernel parameters
	int grid, block;

	// Stores source vertex
	int source;

	int number_of_times_kernel_is_called;

	// Initialize vertices
	vertices[0].index_of_first_adjacent_node_in_edge_array = 0;
	vertices[0].number_of_adjacent_nodes = 2;

	vertices[1].index_of_first_adjacent_node_in_edge_array = 2;
	vertices[1].number_of_adjacent_nodes = 1;

	vertices[2].index_of_first_adjacent_node_in_edge_array = 3;
	vertices[2].number_of_adjacent_nodes = 1;

	vertices[3].index_of_first_adjacent_node_in_edge_array = 4;
	vertices[3].number_of_adjacent_nodes = 1;

	vertices[4].index_of_first_adjacent_node_in_edge_array = 5;
	vertices[4].number_of_adjacent_nodes = 0;

	// Initialize edges
	edges[0] = 1;
	edges[1] = 2;	
	edges[2] = 4;
	edges[3] = 3;
	edges[4] = 4;

	// Create and initialize frontier, visited and cost arrays
	bool frontier[NUMBER_OF_VERTICES] = { false };
	bool visited[NUMBER_OF_VERTICES] = { false };
	int cost[NUMBER_OF_VERTICES] = { 0 };

	// Initialize and insert source vertex into frontier array
	source = 0;
	frontier[source] = true;

	// Allocate device memory for necessary arrays
	hipMalloc((void**)&device_vertices, sizeof(vertex) * NUMBER_OF_VERTICES);
	hipMalloc((void**)&device_edges, sizeof(int) * NUMBER_OF_VERTICES);
	hipMalloc((void**)&device_frontier, sizeof(bool) * NUMBER_OF_VERTICES);
	hipMalloc((void**)&device_visited, sizeof(bool) * NUMBER_OF_VERTICES);
	hipMalloc((void**)&device_cost, sizeof(int) * NUMBER_OF_VERTICES);
	hipMalloc((void**)&device_done, sizeof(bool));

	// Transfer arrays from CPU memory to GPU global memory
	hipMemcpy(device_vertices, vertices, sizeof(vertex) * NUMBER_OF_VERTICES, hipMemcpyHostToDevice);
	hipMemcpy(device_edges, edges, sizeof(int) * NUMBER_OF_VERTICES, hipMemcpyHostToDevice);
	hipMemcpy(device_frontier, frontier, sizeof(bool) * NUMBER_OF_VERTICES, hipMemcpyHostToDevice);
	hipMemcpy(device_visited, visited, sizeof(bool) * NUMBER_OF_VERTICES, hipMemcpyHostToDevice);
	hipMemcpy(device_cost, cost, sizeof(int) * NUMBER_OF_VERTICES, hipMemcpyHostToDevice);

	// Set grid and block sizes
	grid = 1;
	block = 5;

	// Perform level order traversal until all vertices have been visited
	number_of_times_kernel_is_called = 0;
	printf("\n\nOrder: \n\n");
	do {
		number_of_times_kernel_is_called++;
		done = true;
		hipMemcpy(device_done, &done, sizeof(bool), hipMemcpyHostToDevice);
		bfs_gpu <<< grid, block >>> (device_vertices, device_edges, device_frontier, device_visited, device_cost, device_done);
		hipMemcpy(&done, device_done, sizeof(bool), hipMemcpyDeviceToHost);

	} while (!done); 

	// Transfer cost array back to host memory
	hipMemcpy(cost, device_cost, sizeof(int) * NUMBER_OF_VERTICES, hipMemcpyDeviceToHost);
	
	printf("Number of times the kernel is called : %d \n", number_of_times_kernel_is_called);

	printf("\nCost: ");
	for (int i = 0; i < NUMBER_OF_VERTICES; i++)
		printf( "%d    ", cost[i]);
	printf("\n");	

	// Return allocated memory back to device
	hipFree(device_vertices);
	hipFree(device_edges);
	hipFree(device_frontier);
	hipFree(device_visited);
	hipFree(device_cost);
	hipFree(device_done);
}
